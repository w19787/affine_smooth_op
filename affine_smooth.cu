#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>
// #include <math_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <unistd.h>

#define EIGEN_USE_GPU
#include "tensorflow/core/util/gpu_kernel_helper.h"

namespace tensorflow{

#define TB 256
#define EPS 1e-7

__device__ bool InverseMat4x4(double m_in[4][4], double inv_out[4][4]) {
	double m[16], inv[16];
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			m[i * 4 + j] = m_in[i][j];
		}
	}

    inv[0] = m[5]  * m[10] * m[15] -
             m[5]  * m[11] * m[14] -
             m[9]  * m[6]  * m[15] +
             m[9]  * m[7]  * m[14] +
             m[13] * m[6]  * m[11] -
             m[13] * m[7]  * m[10];

    inv[4] = -m[4]  * m[10] * m[15] +
              m[4]  * m[11] * m[14] +
              m[8]  * m[6]  * m[15] -
              m[8]  * m[7]  * m[14] -
              m[12] * m[6]  * m[11] +
              m[12] * m[7]  * m[10];

    inv[8] = m[4]  * m[9] * m[15] -
             m[4]  * m[11] * m[13] -
             m[8]  * m[5] * m[15] +
             m[8]  * m[7] * m[13] +
             m[12] * m[5] * m[11] -
             m[12] * m[7] * m[9];

    inv[12] = -m[4]  * m[9] * m[14] +
               m[4]  * m[10] * m[13] +
               m[8]  * m[5] * m[14] -
               m[8]  * m[6] * m[13] -
               m[12] * m[5] * m[10] +
               m[12] * m[6] * m[9];

    inv[1] = -m[1]  * m[10] * m[15] +
              m[1]  * m[11] * m[14] +
              m[9]  * m[2] * m[15] -
              m[9]  * m[3] * m[14] -
              m[13] * m[2] * m[11] +
              m[13] * m[3] * m[10];

    inv[5] = m[0]  * m[10] * m[15] -
             m[0]  * m[11] * m[14] -
             m[8]  * m[2] * m[15] +
             m[8]  * m[3] * m[14] +
             m[12] * m[2] * m[11] -
             m[12] * m[3] * m[10];

    inv[9] = -m[0]  * m[9] * m[15] +
              m[0]  * m[11] * m[13] +
              m[8]  * m[1] * m[15] -
              m[8]  * m[3] * m[13] -
              m[12] * m[1] * m[11] +
              m[12] * m[3] * m[9];

    inv[13] = m[0]  * m[9] * m[14] -
              m[0]  * m[10] * m[13] -
              m[8]  * m[1] * m[14] +
              m[8]  * m[2] * m[13] +
              m[12] * m[1] * m[10] -
              m[12] * m[2] * m[9];

    inv[2] = m[1]  * m[6] * m[15] -
             m[1]  * m[7] * m[14] -
             m[5]  * m[2] * m[15] +
             m[5]  * m[3] * m[14] +
             m[13] * m[2] * m[7] -
             m[13] * m[3] * m[6];

    inv[6] = -m[0]  * m[6] * m[15] +
              m[0]  * m[7] * m[14] +
              m[4]  * m[2] * m[15] -
              m[4]  * m[3] * m[14] -
              m[12] * m[2] * m[7] +
              m[12] * m[3] * m[6];

    inv[10] = m[0]  * m[5] * m[15] -
              m[0]  * m[7] * m[13] -
              m[4]  * m[1] * m[15] +
              m[4]  * m[3] * m[13] +
              m[12] * m[1] * m[7] -
              m[12] * m[3] * m[5];

    inv[14] = -m[0]  * m[5] * m[14] +
               m[0]  * m[6] * m[13] +
               m[4]  * m[1] * m[14] -
               m[4]  * m[2] * m[13] -
               m[12] * m[1] * m[6] +
               m[12] * m[2] * m[5];

    inv[3] = -m[1] * m[6] * m[11] +
              m[1] * m[7] * m[10] +
              m[5] * m[2] * m[11] -
              m[5] * m[3] * m[10] -
              m[9] * m[2] * m[7] +
              m[9] * m[3] * m[6];

    inv[7] = m[0] * m[6] * m[11] -
             m[0] * m[7] * m[10] -
             m[4] * m[2] * m[11] +
             m[4] * m[3] * m[10] +
             m[8] * m[2] * m[7] -
             m[8] * m[3] * m[6];

    inv[11] = -m[0] * m[5] * m[11] +
               m[0] * m[7] * m[9] +
               m[4] * m[1] * m[11] -
               m[4] * m[3] * m[9] -
               m[8] * m[1] * m[7] +
               m[8] * m[3] * m[5];

    inv[15] = m[0] * m[5] * m[10] -
              m[0] * m[6] * m[9] -
              m[4] * m[1] * m[10] +
              m[4] * m[2] * m[9] +
              m[8] * m[1] * m[6] -
              m[8] * m[2] * m[5];

    double det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

    if (abs(det) < 1e-9) {
        return false;
    }


    det = 1.0 / det;

    for (int i = 0; i < 4; i++) {
    	for (int j = 0; j < 4; j++) {
    		inv_out[i][j] = inv[i * 4 + j] * det;
    	}
    }

    return true;
}

__global__ void best_local_affine_kernel(
	const float *output, const float *input, float *affine_model,
	int h, int w, float epsilon, int kernel_radius
)
{
	int size = h * w;
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int x = id % w, y = id / w;

		double Mt_M[4][4] = {}; // 4x4
		double invMt_M[4][4] = {};
		double Mt_S[3][4] = {}; // RGB -> 1x4
		double A[3][4] = {};
		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 4; j++) {
				Mt_M[i][j] = 0, invMt_M[i][j] = 0;
				if (i != 3) {
					Mt_S[i][j] = 0, A[i][j] = 0;
					if (i == j)
			    		Mt_M[i][j] = 1e-3;
			    }
			}

		for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
			for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {

				int xx = x + dx, yy = y + dy;
				int id2 = yy * w + xx;

				if (0 <= xx && xx < w && 0 <= yy && yy < h) {

					Mt_M[0][0] += input[id2 + 2*size] * input[id2 + 2*size];
					Mt_M[0][1] += input[id2 + 2*size] * input[id2 + size];
					Mt_M[0][2] += input[id2 + 2*size] * input[id2];
					Mt_M[0][3] += input[id2 + 2*size];

					Mt_M[1][0] += input[id2 + size] * input[id2 + 2*size];
					Mt_M[1][1] += input[id2 + size] * input[id2 + size];
					Mt_M[1][2] += input[id2 + size] * input[id2];
					Mt_M[1][3] += input[id2 + size];

					Mt_M[2][0] += input[id2] * input[id2 + 2*size];
					Mt_M[2][1] += input[id2] * input[id2 + size];
					Mt_M[2][2] += input[id2] * input[id2];
					Mt_M[2][3] += input[id2];

					Mt_M[3][0] += input[id2 + 2*size];
					Mt_M[3][1] += input[id2 + size];
					Mt_M[3][2] += input[id2];
					Mt_M[3][3] += 1;

					Mt_S[0][0] += input[id2 + 2*size] * output[id2 + 2*size];
					Mt_S[0][1] += input[id2 + size] * output[id2 + 2*size];
					Mt_S[0][2] += input[id2] * output[id2 + 2*size];
					Mt_S[0][3] += output[id2 + 2*size];

					Mt_S[1][0] += input[id2 + 2*size] * output[id2 + size];
					Mt_S[1][1] += input[id2 + size] * output[id2 + size];
					Mt_S[1][2] += input[id2] * output[id2 + size];
					Mt_S[1][3] += output[id2 + size];

					Mt_S[2][0] += input[id2 + 2*size] * output[id2];
					Mt_S[2][1] += input[id2 + size] * output[id2];
					Mt_S[2][2] += input[id2] * output[id2];
					Mt_S[2][3] += output[id2];
				}
			}
		}

		bool success = InverseMat4x4(Mt_M, invMt_M);

		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 4; j++) {
				for (int k = 0; k < 4; k++) {
					A[i][j] += invMt_M[j][k] * Mt_S[i][k];
				}
			}
		}

		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 4; j++) {
				int affine_id = i * 4 + j;
				affine_model[12 * id + affine_id] = A[i][j];
			}
		}



	}
	return ;
}

__global__ void bilateral_smooth_kernel(
	float *affine_model, float *filtered_affine_model, const float *guide,
	int h, int w, int kernel_radius, float sigma1, float sigma2
)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	if (id < size) {
		int x = id % w;
		int y = id / w;

		double sum_affine[12] = {};
		double sum_weight = 0;
		for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
			for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
				int yy = y + dy, xx = x + dx;
				int id2 = yy * w + xx;
				if (0 <= xx && xx < w && 0 <= yy && yy < h) {
					float color_diff1 = guide[yy*w + xx] - guide[y*w + x];
					float color_diff2 = guide[yy*w + xx + size] - guide[y*w + x + size];
					float color_diff3 = guide[yy*w + xx + 2*size] - guide[y*w + x + 2*size];
					float color_diff_sqr =
						(color_diff1*color_diff1 + color_diff2*color_diff2 + color_diff3*color_diff3) / 3;

					float v1 = exp(-(dx * dx + dy * dy) / (2 * sigma1 * sigma1));
					float v2 = exp(-(color_diff_sqr) / (2 * sigma2 * sigma2));
					float weight = v1 * v2;

					for (int i = 0; i < 3; i++) {
						for (int j = 0; j < 4; j++) {
							int affine_id = i * 4 + j;
							sum_affine[affine_id] += weight * affine_model[id2*12 + affine_id];
						}
					}
					sum_weight += weight;
				}
			}
		}

		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 4; j++) {
				int affine_id = i * 4 + j;
				filtered_affine_model[id*12 + affine_id] = sum_affine[affine_id] / sum_weight;
			}
		}
	}
	return ;
}

__global__ void reconstruction_best_kernel(
	const float *input, float *filtered_affine_model, float *filtered_best_output,
	int h, int w
)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	if (id < size) {
		double out1 =
			input[id + 2*size] * filtered_affine_model[id*12 + 0] + // A[0][0] +
			input[id + size]   * filtered_affine_model[id*12 + 1] + // A[0][1] +
			input[id]          * filtered_affine_model[id*12 + 2] + // A[0][2] +
								 filtered_affine_model[id*12 + 3]; //A[0][3];
		double out2 =
			input[id + 2*size] * filtered_affine_model[id*12 + 4] + //A[1][0] +
			input[id + size]   * filtered_affine_model[id*12 + 5] + //A[1][1] +
			input[id]          * filtered_affine_model[id*12 + 6] + //A[1][2] +
								 filtered_affine_model[id*12 + 7]; //A[1][3];
		double out3 =
			input[id + 2*size] * filtered_affine_model[id*12 + 8] + //A[2][0] +
			input[id + size]   * filtered_affine_model[id*12 + 9] + //A[2][1] +
			input[id]          * filtered_affine_model[id*12 + 10] + //A[2][2] +
								 filtered_affine_model[id*12 + 11]; // A[2][3];

		filtered_best_output[id] = out1;
		filtered_best_output[id + size] = out2;
		filtered_best_output[id + 2*size] = out3;
	}
	return ;
}

// __global__ void apply_affine_smooth(const float* output, const float* input, 
// 	float epsilon, int patch, int h, int w, float f_r, float f_e, float* smooth_output)
// {

// 	// hipMallocManaged(affine_model, h*w*12*sizeof(float));
// 	// hipMallocManaged(filtered_affine_model, h*w*12*sizeof(float));
// 	best_local_affine_kernel(output, input, affine_model, h, w, epsilon, radius);
// 	bilateral_smooth_kernel(affine_model, filtered_affine_model, input, h, w, radius, sigma1, sigma2);
// 	reconstruction_best_kernel(input, filtered_affine_model, smooth_output, h, w);

// }


void AffineSmoothKernalLauncher(const float* output, const float* input, const float* p_epsilon, const int* p_patch, 
	const int* ph, const int* pw, const float* pf_r, const float* pf_e, float* output_affine, int block_count, int threads_per_block, hipStream_t stream)
{
	float epsilon = ldg(p_epsilon);
	int patch = ldg(p_patch);
	int h = ldg(ph);
	int w = ldg(pw);
	int f_r = ldg(pf_r);
	int f_e = ldg(pf_e);

	float sigma1 = f_r/3;
	float sigma2 = f_e;

	float* affine_model;
	float* filtered_affine_model;

	int radius = (patch - 1) / 2;

	affine_model = (float *)malloc(h*w*12*sizeof(float));
	filtered_affine_model = (float *)malloc(h*w*12*sizeof(float));

	best_local_affine_kernel<<<block_count, threads_per_block, 0, stream>>>(output, input, affine_model, h, w, epsilon, radius);
	bilateral_smooth_kernel<<<block_count, threads_per_block, 0, stream>>>(affine_model, filtered_affine_model, input, h, w, radius, sigma1, sigma2);
	reconstruction_best_kernel<<<block_count, threads_per_block, 0, stream>>>(input, filtered_affine_model, output_affine, h, w);

	free(affine_model);
	free(filtered_affine_model);
}

}